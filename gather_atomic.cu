#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdlib>
#include <sys/time.h>

#include "src/gather_atomic.cuh"
#include "src/parse_data.h"

#define METHOD 2

#define CUDA_RT_CALL(call)                                                              \
{                                                                                       \
    hipError_t cudaStatus = call;                                                      \
    if (hipSuccess != cudaStatus)                                                      \
        fprintf(stderr,                                                                 \
                "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                "with "                                                                 \
                "%s (%d).\n",                                                           \
                #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
}

struct timeval t1, t2;
double mytime; 

int main(void) {
    int *input_row_ind_d, *input_col_ind_d, *input_csr_row_d;
    int *input_row_ind_h, *input_col_ind_h, *input_csr_row_h;
    int *total_triangle_d, *total_triangle_h;

    // char file[] = "./data/Theory-3-4-B1k.tsv";
    // int num_nodes = 20;
    // int num_edges = 31;
    
    // char file[] = "./data/Theory-25-81-B1k.tsv";
    // int num_nodes = 2132;
    // int num_edges = 4156;

    // 133321
    char file[] = "./data/Theory-16-25-81-B1k.tsv";
    int num_nodes = 36244;
    int num_edges = 137164;

    // 2102761
    // char file[] = "./data/Theory-25-81-256-B1k.tsv";
    // int num_nodes = 547924;
    // int num_edges = 2132284;
    
    // 66758995
    // char file[] = "./data/Theory-5-9-16-25-81-B1k.tsv";
    // int num_nodes = 2174640;
    // int num_edges = 28667380;

    input_row_ind_h = (int*) malloc(num_edges * sizeof(int));
    input_col_ind_h = (int*) malloc(num_edges * sizeof(int));
    input_csr_row_h = (int*) malloc((num_nodes + 1) * sizeof(int));
    total_triangle_h = (int*) malloc(sizeof(int));

    CUDA_RT_CALL( hipMalloc((void **)&input_row_ind_d, num_edges * sizeof(int)) );
    CUDA_RT_CALL( hipMalloc((void **)&input_col_ind_d, num_edges * sizeof(int)) );
    CUDA_RT_CALL( hipMalloc((void **)&input_csr_row_d, (num_nodes + 1) * sizeof(int)) );
    CUDA_RT_CALL( hipMalloc((void **)&total_triangle_d, sizeof(int)));

    read_tsv(input_row_ind_h, input_col_ind_h, input_csr_row_h, num_nodes + 1, num_edges, file);

    CUDA_RT_CALL( hipMemcpy(input_row_ind_d, input_row_ind_h, num_edges * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_RT_CALL( hipMemcpy(input_col_ind_d, input_col_ind_h, num_edges * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_RT_CALL( hipMemcpy(input_csr_row_d, input_csr_row_h, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice) );

    int num_blocks = ceil((num_edges * 1.0) / BLOCK_SIZE);

    gettimeofday(&t1, 0);

#if METHOD == 0
    gather_atomic_naive<<<num_blocks,BLOCK_SIZE>>>(num_edges, input_row_ind_d, input_col_ind_d, input_csr_row_d, total_triangle_d);
#elif METHOD == 1
    gather_atomic_block<<<num_blocks,BLOCK_SIZE>>>(num_edges, input_row_ind_d, input_col_ind_d, input_csr_row_d, total_triangle_d);
#elif METHOD == 2
    gather_atomic_warp<<<num_blocks,BLOCK_SIZE>>>(num_edges, input_row_ind_d, input_col_ind_d, input_csr_row_d, total_triangle_d);
#endif 

    CUDA_RT_CALL( hipDeviceSynchronize() ); 

    gettimeofday(&t2, 0);
    mytime = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    
    printf("Time elapsed: %3.5f ms\n", mytime); 

    CUDA_RT_CALL( hipMemcpy(total_triangle_h, total_triangle_d, sizeof(int), hipMemcpyDeviceToHost) );
    printf("Total number of triangle: %d\n", total_triangle_h[0]);

    CUDA_RT_CALL( hipFree(input_row_ind_d) );
    CUDA_RT_CALL( hipFree(input_col_ind_d) );
    CUDA_RT_CALL( hipFree(input_csr_row_d) );
    CUDA_RT_CALL( hipFree(total_triangle_d) );

    free(input_row_ind_h);
    free(input_col_ind_h);
    free(input_csr_row_h);
    free(total_triangle_h);

    return 0;
}
